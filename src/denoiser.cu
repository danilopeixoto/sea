// Copyright (c) 2020, Danilo Peixoto. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sea/denoiser.h>
#include <sea/renderer.h>

#include <optix_function_table_definition.h>

SEA_NAMESPACE_BEGIN

__host__ Denoiser * denoiser_create(DenoiserType type) {
	Denoiser * denoiser = new Denoiser;

	denoiser->type = type;

	denoiser->cuda_context = 0;
	denoiser->options = {};

	denoiser->denoiser_options.inputKind = OPTIX_DENOISER_INPUT_RGB_ALBEDO_NORMAL;
	denoiser->denoiser_options.pixelFormat = OPTIX_PIXEL_FORMAT_FLOAT3;

	denoiser->state_size = 0;
	denoiser->scratch_size = 0;

	denoiser->state = 0;

	switch (type) {
	case HDR:
		denoiser->model = OPTIX_DENOISER_MODEL_KIND_HDR;
		break;
	default:
		denoiser->model = OPTIX_DENOISER_MODEL_KIND_LDR;
	}

	hipFree(0);
	optixInit();

	optixDeviceContextCreate(denoiser->cuda_context, &denoiser->options, &denoiser->context);
	optixDenoiserCreate(denoiser->context, &denoiser->denoiser_options, &denoiser->denoiser_handler);
	optixDenoiserSetModel(denoiser->denoiser_handler, denoiser->model, nullptr, 0);

	return denoiser;
}
__host__ void denoiser_update(Denoiser * denoiser, const Renderer * renderer) {
	unsigned int width = renderer->image->width;
	unsigned int height = renderer->image->height;

	OptixDenoiserSizes sizes;
	optixDenoiserComputeMemoryResources(denoiser->denoiser_handler, width, height, &sizes);

	hipStream_t stream;
	hipStreamCreate(&stream);

	denoiser->state_size = sizes.stateSizeInBytes;
	denoiser->scratch_size = sizes.recommendedScratchSizeInBytes;

	if (denoiser->state)
		hipFree((void *)denoiser->state);

	hipMalloc((void **)&denoiser->state, denoiser->state_size + denoiser->scratch_size);

	optixDenoiserSetup(
		denoiser->denoiser_handler, stream, width, height,
		denoiser->state, denoiser->state_size,
		denoiser->state + denoiser->state_size, denoiser->scratch_size);

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);
}
__host__ void denoiser_delete(Denoiser * denoiser) {
	if (denoiser) {
		optixDenoiserDestroy(denoiser->denoiser_handler);
		optixDeviceContextDestroy(denoiser->context);

		if (denoiser->state)
			hipFree((void *)denoiser->state);

		delete denoiser;

		denoiser = nullptr;
	}
}

__host__ void denoiser_denoise(Denoiser * denoiser, Renderer * renderer, float blend_factor) {
	unsigned int width = renderer->image->width;
	unsigned int height = renderer->image->height;

	Image * passes[4] = {
		renderer->frame_radiance_pass,
		renderer->albedo_pass,
		renderer->normal_pass,
		renderer->frame_radiance_pass };

	hipStream_t stream;
	hipStreamCreate(&stream);

	OptixDenoiserParams parameters = {};
	parameters.blendFactor = 1.0f - blend_factor;

	OptixImage2D layers[4];

	for (size_t i = 0; i < 4; i++) {
		OptixImage2D & layer = layers[i];

		layer.width = width;
		layer.height = height;
		layer.pixelStrideInBytes = 3 * sizeof(float);
		layer.rowStrideInBytes = layer.pixelStrideInBytes * width;
		layer.format = OPTIX_PIXEL_FORMAT_FLOAT3;

		layer.data = (hipDeviceptr_t)passes[i]->data;
	}

	OptixImage2D * input_layers = layers;
	OptixImage2D * output_layer = layers + 3;

	optixDenoiserInvoke(
		denoiser->denoiser_handler, stream,
		&parameters, denoiser->state, denoiser->state_size, input_layers, 3, 0, 0,
		output_layer, denoiser->state + denoiser->state_size, denoiser->scratch_size);

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);
}

SEA_NAMESPACE_END