#include "hip/hip_runtime.h"
// Copyright (c) 2020, Danilo Peixoto. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
//
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
//
// * Neither the name of the copyright holder nor the names of its
//   contributors may be used to endorse or promote products derived from
//   this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sea/renderer.h>
#include <sea/utility.h>
#include <sea/scene.h>
#include <sea/math.h>

#include <glm/trigonometric.hpp>
#include <glm/vec2.hpp>
#include <glm/common.hpp>

#include <cmath>

SEA_NAMESPACE_BEGIN

__host__ Image * image_create(size_t width, size_t height) {
    Image * image;
    hipMallocManaged(&image, sizeof(Image));

    image->width = width;
    image->height = height;

    hipMallocManaged(&image->data, width * height * 3 * sizeof(float));

    return image;
}
__host__ void image_delete(Image * image) {
    if (image) {
        hipFree(image->data);
        hipFree(image);

        image = nullptr;
    }
}
__host__ __device__ void image_set(
    Image * image, size_t x, size_t y, const glm::vec3 & color) {
    size_t index = (x + y * image->width) * 3;

    image->data[index] = color.r;
    image->data[index + 1] = color.g;
    image->data[index + 2] = color.b;
}
__host__ __device__ void image_get(
    const Image * image, size_t x, size_t y, glm::vec3 & color) {
    size_t index = (x + y * image->width) * 3;

    color.r = image->data[index];
    color.g = image->data[index + 1];
    color.b = image->data[index + 2];
}

__host__ Renderer * renderer_create(size_t width, size_t height,
    size_t camera_samples, size_t light_samples, size_t maximum_depth, float gamma) {
    Renderer * renderer;
    hipMallocManaged(&renderer, sizeof(Renderer));

    renderer->camera_samples = camera_samples;
    renderer->light_samples = light_samples;
    renderer->maximum_depth = maximum_depth;
	renderer->pass = 0;
    renderer->time = 0;
    renderer->progressive_time = 0;
    renderer->accumulated = 0.0f;

    renderer->gamma = gamma;

	renderer->frame_radiance_pass = image_create(width, height);
    renderer->radiance_pass = image_create(width, height);
	renderer->albedo_pass = image_create(width, height);
	renderer->normal_pass = image_create(width, height);

	renderer->image = renderer->radiance_pass;

    return renderer;
}
__host__ void renderer_update(Renderer * renderer, Camera * camera, size_t width, size_t height) {
	camera->film.width = (float)width;
    camera->film.height = (float)height;

	image_delete(renderer->frame_radiance_pass);
    image_delete(renderer->radiance_pass);
	image_delete(renderer->albedo_pass);
	image_delete(renderer->normal_pass);

	renderer->frame_radiance_pass = image_create(width, height);
	renderer->radiance_pass = image_create(width, height);
	renderer->albedo_pass = image_create(width, height);
	renderer->normal_pass = image_create(width, height);

	Image * passes[3] = {
		renderer->radiance_pass,
		renderer->albedo_pass,
		renderer->normal_pass };

	renderer->image = passes[renderer->pass];
}
__host__ void renderer_delete(Renderer * renderer) {
    if (renderer) {
		image_delete(renderer->frame_radiance_pass);
        image_delete(renderer->radiance_pass);
		image_delete(renderer->albedo_pass);
		image_delete(renderer->normal_pass);

        hipFree(renderer);

        renderer = nullptr;
    }
}
__global__ void renderer_initialize(const Renderer * renderer, hiprandState * random_state) {
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < renderer->image->width && j < renderer->image->height) {
		size_t index = i + j * renderer->image->width;
		hiprand_init(renderer->time, index, 0, &random_state[index]);
	}
}
__global__ void renderer_render_passes(const Scene * scene, Camera * camera, Renderer * renderer, hiprandState * random_state) {
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < renderer->image->width && j < renderer->image->height) {
		size_t index = i + j * renderer->image->width;
		hiprandState local_random_state = random_state[index];

		glm::vec3 output_albedo(0.0f, 0.0f, 0.0f);
		glm::vec3 output_normal(0.0f, 0.0f, 0.0f);

		for (size_t k = 0; k < renderer->camera_samples; k++) {
			Ray camera_ray;
			camera_generate_ray(camera, (float)i, (float)j, local_random_state, camera_ray);

			glm::vec3 albedo, normal;
			renderer_trace_passes(scene, camera, renderer, camera_ray, local_random_state, albedo, normal);

			output_albedo += albedo;
			output_normal += normal;
		}

		output_albedo /= (float)renderer->camera_samples;
		output_normal /= (float)renderer->camera_samples;

		color_gamma(output_albedo, renderer->gamma);
		color_saturate(output_albedo);

		glm::vec3 previous_albedo, previous_normal;

		image_get(renderer->albedo_pass, i, j, previous_albedo);
		image_get(renderer->normal_pass, i, j, previous_normal);

		output_albedo = color_mix(output_albedo, previous_albedo, renderer->accumulated);
		output_normal = color_mix(output_normal, previous_normal, renderer->accumulated);

		image_set(renderer->albedo_pass, i, j, output_albedo);
		image_set(renderer->normal_pass, i, j, output_normal);
	}
}
__device__ void renderer_trace_passes(
	const Scene * scene, const Camera * camera, const Renderer * renderer, const Ray & ray, hiprandState & state,
	glm::vec3 & albedo, glm::vec3 & normal) {
	Intersection intersection;

	if (!scene_intersects_ray(scene, ray, intersection)) {
		albedo = glm::vec3(0.0f, 0.0f, 0.0f);
		normal = glm::vec3(0.0f, 0.0f, 0.0f);
	}
	else {
		Triangle * triangle = scene->triangle_list[intersection.index];
		BSDF * bsdf = triangle->bsdf;

		ShaderGlobals shader_globals;
		triangle_shader_globals(triangle, ray, intersection, shader_globals);

		albedo = SEA_INV_PI * bsdf->intensity * bsdf->color;
		normal = glm::inverse(camera->view_matrix) * glm::vec4(shader_globals.normal, 0.0f);
	}
}
__global__ void renderer_render(const Scene * scene, Camera * camera, Renderer * renderer, hiprandState * random_state) {
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < renderer->image->width && j < renderer->image->height) {
		size_t index = i + j * renderer->image->width;
		hiprandState local_random_state = random_state[index];

		Ray camera_ray;
		camera_generate_ray(camera, (float)i, (float)j, local_random_state, camera_ray);

		glm::vec3 radiance = renderer_trace(scene, camera, renderer, camera_ray, local_random_state);

		color_gamma(radiance, renderer->gamma);
		color_saturate(radiance);

		image_set(renderer->frame_radiance_pass, i, j, radiance);
	}
}
__device__ glm::vec3 renderer_trace(
	const Scene * scene, const Camera * camera, const Renderer * renderer, const Ray & ray, hiprandState & state) {
    glm::vec3 radiance(0.0f, 0.0f, 0.0f);
    glm::vec3 weight(1.0f, 1.0f, 1.0f);

    Ray temp_ray = ray;

    for (size_t depth = 0; depth < renderer->maximum_depth; depth++) {
        Intersection intersection;

        if (!scene_intersects_ray(scene, temp_ray, intersection))
            break;

        Triangle * triangle = scene->triangle_list[intersection.index];
        BSDF * bsdf = triangle->bsdf;

        if (bsdf->type == BSDF_LIGHT) {
            radiance += color_multiply(weight, bsdf->intensity * bsdf->color);
            break;
        }
        else {
			ShaderGlobals shader_globals;
			triangle_shader_globals(triangle, temp_ray, intersection, shader_globals);

            glm::vec3 direct_light(0.0f, 0.0f, 0.0f);

            for (size_t j = 0; j < renderer->light_samples; j++) {
				size_t light_index = random_uniform_uinteger(state) % scene->light_count;
				Triangle * light = scene->light_group[light_index];

                const glm::vec3 & v0 = light->vertices.v0.position;
                const glm::vec3 & v1 = light->vertices.v1.position;
                const glm::vec3 & v2 = light->vertices.v2.position;

                glm::vec3 uvw = random_uniform_triangle(random_uniform_2D(state));
                glm::vec3 point = uvw.x * v0 + uvw.y * v1 + uvw.z * v2;

                Ray shadow_ray;
                shadow_ray.origin = shader_globals.point + SEA_BIAS * shader_globals.normal;
                shadow_ray.direction = point - shader_globals.point;

                float squared_light_distance = glm::dot(shadow_ray.direction, shadow_ray.direction);
                shadow_ray.direction /= glm::sqrt(squared_light_distance);

                Intersection shadow_intersection;
                scene_intersects_ray(scene, shadow_ray, shadow_intersection);

                if (shadow_intersection.hit && scene->triangle_list[shadow_intersection.index] == light) {
					ShaderGlobals light_shader_globals;
                    triangle_shader_globals(light, shadow_ray, shadow_intersection, light_shader_globals);

                    glm::vec3 diffuse_bsdf = SEA_INV_PI * bsdf->intensity * bsdf->color;
                    float cosine = glm::max(0.0f, glm::dot(shader_globals.normal, shadow_ray.direction));

                    glm::vec3 light_emission = light->bsdf->intensity * light->bsdf->color;
                    float light_inverse_pdf = light->surface_area;
                    float light_cosine = glm::max(0.0f, glm::dot(light_shader_globals.normal, -shadow_ray.direction));

                    glm::vec3 estimated_light = light_emission * light_inverse_pdf * light_cosine / squared_light_distance;

                    direct_light += color_multiply(diffuse_bsdf, estimated_light) * cosine;
                }
            }

            direct_light *= (float)scene->light_count / (float)renderer->light_samples;
            radiance += color_multiply(weight, direct_light);

            temp_ray.origin = shader_globals.point + SEA_BIAS * shader_globals.normal;

            glm::vec3 sample = random_uniform_cosine_weighted_hemisphere(random_uniform_2D(state));
            world_coordinate_system(sample, shader_globals, temp_ray.direction);

            weight = color_multiply(weight, bsdf->intensity * bsdf->color);

            float p = glm::max(weight.r, glm::max(weight.g, weight.b));

            if (random_uniform_1D(state) > p)
                break;

            weight /= p;
        }
    }

	return radiance;
}
__global__ void renderer_accumulate(Renderer * renderer) {
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < renderer->image->width && j < renderer->image->height) {
		glm::vec3 previous_radiance;
		image_get(renderer->radiance_pass, i, j, previous_radiance);

		glm::vec3 radiance;
		image_get(renderer->frame_radiance_pass, i, j, radiance);

		radiance = color_mix(radiance, previous_radiance, renderer->accumulated);
		image_set(renderer->radiance_pass, i, j, radiance);
	}
}

SEA_NAMESPACE_END